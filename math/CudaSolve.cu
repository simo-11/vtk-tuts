/*
 * Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <cassert>
#include <cstring>
#include <iostream>
#include <string>

#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include "cusolver_utils.h"

int cuda_solve(int verbose, int n, float* a, float* b, int*) {
    // Numer of right hand sides
    const int nrhs = 1;

    // Use float matrix and half precision factorization
    typedef float T;
    const cusolverPrecType_t matrix_precision = CUSOLVER_R_32F;
    // make sure that you specify matrix precision that matches to the data type
    assert(traits<T>::cusolver_precision_type == matrix_precision);
    const cusolverPrecType_t compute_lower_precision = CUSOLVER_R_16F;

    // Use GMRES refinement solver
    const cusolverIRSRefinement_t refinement_solver = CUSOLVER_IRS_REFINE_GMRES;

    T *hA=a;
    int lda=n;
    T *hB=b;
    int ldb=n;
    T *hX=(T*)malloc(n*n*sizeof(T));
    int ldx=n;

    hipStream_t stream;
    hipEvent_t event_start, event_end;
    hipsolverHandle_t handle;
    cusolverDnIRSParams_t gesv_params;
    cusolverDnIRSInfos_t gesv_info;
    if (verbose) {
        std::cout << "Initializing CUDA..." << std::endl;
    }
    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    CUDA_CHECK(hipEventCreate(&event_start));
    CUDA_CHECK(hipEventCreate(&event_end));
    CUSOLVER_CHECK(hipsolverDnCreate(&handle));
    CUSOLVER_CHECK(hipsolverSetStream(handle, stream));

    if (verbose) {
        std::cout << "Setting up gesv() parameters..." << std::endl;
    }
    // create solver parameters
    CUSOLVER_CHECK(cusolverDnIRSParamsCreate(&gesv_params));
    // set matrix precision and factorization precision
    CUSOLVER_CHECK(cusolverDnIRSParamsSetSolverPrecisions(gesv_params, matrix_precision,
                                                          compute_lower_precision));
    // set refinement solver
    CUSOLVER_CHECK(cusolverDnIRSParamsSetRefinementSolver(gesv_params, refinement_solver));
    // create solve info structure
    CUSOLVER_CHECK(cusolverDnIRSInfosCreate(&gesv_info));

    // matrix on device
    T *dA;
    int ldda = ALIGN_TO(n * sizeof(T), device_alignment) / sizeof(T);
    // right hand side on device
    T *dB;
    int lddb = ALIGN_TO(n * sizeof(T), device_alignment) / sizeof(T);
    // solution on device
    T *dX;
    int lddx = ALIGN_TO(n * sizeof(T), device_alignment) / sizeof(T);
    // info indicator on device
    int *dinfo;
    // work buffer
    void *dwork;
    // size of work buffer
    size_t dwork_size;
    // number of refinement iterations returned by solver
    int iter;
    if (verbose) {
        std::cout << "Allocating memory on device..." << std::endl;
    }
    // allocate data
    CUDA_CHECK(hipMalloc(&dA, ldda * n * sizeof(T)));
    CUDA_CHECK(hipMalloc(&dB, lddb * nrhs * sizeof(T)));
    CUDA_CHECK(hipMalloc(&dX, lddx * nrhs * sizeof(T)));
    CUDA_CHECK(hipMalloc(&dinfo, sizeof(int)));

    // copy input data
    size_t dpitch = ldda * sizeof(T);
    size_t spitch = lda * sizeof(T);
    size_t width = n * sizeof(T);
    size_t height = n;
    CUDA_CHECK(hipMemcpy2D(dA, dpitch, hA, spitch, width, height,
        hipMemcpyDefault));
    dpitch = lddb * sizeof(T);
    spitch = ldb * sizeof(T);
    height = nrhs;
    CUDA_CHECK(hipMemcpy2D(dB, dpitch, hB, spitch, width,height,
        hipMemcpyDefault));

    // get required device work buffer size
    CUSOLVER_CHECK(cusolverDnIRSXgesv_bufferSize(handle, gesv_params, n, nrhs, &dwork_size));
    if (verbose) {
        std::cout << "Workspace is " << dwork_size << " bytes" << std::endl;
    }
    CUDA_CHECK(hipMalloc(&dwork, dwork_size));

    if (verbose) {
        std::cout << "Solving matrix on device..." << std::endl;
    }
    CUDA_CHECK(hipEventRecord(event_start, stream));

    hipsolverStatus_t gesv_status =
        cusolverDnIRSXgesv(handle, gesv_params, gesv_info, n, nrhs, dA, ldda, dB, lddb, dX, lddx,
                           dwork, dwork_size, &iter, dinfo);
    CUSOLVER_CHECK(gesv_status);

    CUDA_CHECK(hipEventRecord(event_end, stream));
    // check solve status
    int info = 0;
    CUDA_CHECK(
        hipMemcpyAsync(&info, dinfo, sizeof(int), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK(hipStreamSynchronize(stream));

    if (verbose) {
        std::cout << "Solve info is: " << info << ", iter is: " << iter << std::endl;
    }

    CUDA_CHECK(hipMemcpy2D(hX, ldx * sizeof(T), dX, lddx * sizeof(T), n * sizeof(T), nrhs,
                            hipMemcpyDefault));
    CUDA_CHECK(hipGetLastError());
    memcpy(b, hX, n * sizeof(T));
    float solve_time = 0.f;
    CUDA_CHECK(hipEventElapsedTime(&solve_time, event_start, event_end));
    if (verbose) {
        std::cout << "Solved matrix " << n << "x" << n << " with " << nrhs << " right hand sides in "
            << solve_time << " ms" << std::endl;
    }

    if (verbose) {
        std::cout << "Releasing resources..." << std::endl;
    }
    CUDA_CHECK(hipFree(dwork));
    CUDA_CHECK(hipFree(dinfo));
    CUDA_CHECK(hipFree(dX));
    CUDA_CHECK(hipFree(dB));
    CUDA_CHECK(hipFree(dA));
    CUSOLVER_CHECK(hipsolverDnDestroy(handle));
    CUDA_CHECK(hipEventDestroy(event_start));
    CUDA_CHECK(hipEventDestroy(event_end));
    CUDA_CHECK(hipStreamDestroy(stream));
    free(hX);
    if (verbose) {
        std::cout << "Done!" << std::endl;
    }
    return 0;
}
