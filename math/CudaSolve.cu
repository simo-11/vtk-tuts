#include <cassert>
#include <cstring>
#include <iostream>
#include <string>

#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include "cusolver_utils.h"
#include "MathExample.h"
enum errorCode { HX_ALLOCATE = 1 };
int cuda_allocate(Workspaces* ws) {
    int n = ws->n;
    typedef float T;
    ws->hx= (T*)malloc(n * n * sizeof(T));
    if (ws->hx == nullptr) {
        return HX_ALLOCATE;
    }
    const int nrhs = 1;
    const cusolverPrecType_t matrix_precision = CUSOLVER_R_32F;
    // make sure that you specify matrix precision that matches to the data type
    assert(traits<T>::cusolver_precision_type == matrix_precision);
    const cusolverPrecType_t compute_lower_precision = CUSOLVER_R_16F;
    // Use GMRES refinement solver
    const cusolverIRSRefinement_t refinement_solver = CUSOLVER_IRS_REFINE_GMRES;
    CUDA_CHECK(hipStreamCreateWithFlags(&ws->stream, hipStreamNonBlocking)); // 112 ms
    CUSOLVER_CHECK(hipsolverDnCreate(&ws->handle)); // 62 ms
    CUSOLVER_CHECK(hipsolverSetStream(ws->handle, ws->stream));
    // create solver parameters
    CUSOLVER_CHECK(cusolverDnIRSParamsCreate(&ws->gesv_params));
    // set matrix precision and factorization precision
    CUSOLVER_CHECK(cusolverDnIRSParamsSetSolverPrecisions
        (ws->gesv_params, matrix_precision,
        compute_lower_precision));
    // set refinement solver
    CUSOLVER_CHECK(cusolverDnIRSParamsSetRefinementSolver
        (ws->gesv_params, refinement_solver));
    // create solve info structure
    CUSOLVER_CHECK(cusolverDnIRSInfosCreate(&ws->gesv_info));
    ws->ldda = ALIGN_TO(n * sizeof(T), device_alignment) / sizeof(T);
    ws->lddb = ALIGN_TO(n * sizeof(T), device_alignment) / sizeof(T);
    ws->lddx = ALIGN_TO(n * sizeof(T), device_alignment) / sizeof(T);
    CUDA_CHECK(hipMalloc(&ws->dA, ws->ldda * n * sizeof(T)));
    CUDA_CHECK(hipMalloc(&ws->dB, ws->lddb * nrhs * sizeof(T)));
    CUDA_CHECK(hipMalloc(&ws->dX, ws->lddx * nrhs * sizeof(T)));
    CUDA_CHECK(hipMalloc(&ws->dinfo, sizeof(int)));
    // get required device work buffer size
    CUSOLVER_CHECK(cusolverDnIRSXgesv_bufferSize(ws->handle, ws->gesv_params, 
        n, nrhs, &ws->dwork_size));
    CUDA_CHECK(hipMalloc(&ws->dwork, ws->dwork_size));
    return 0;
}
int cuda_free(Workspaces* ws) {
    if (ws->hx != nullptr) {
        free(ws->hx);
    }
    ws->hx = nullptr;
    CUDA_CHECK(hipFree(ws->dwork));
    CUDA_CHECK(hipFree(ws->dinfo));
    CUDA_CHECK(hipFree(ws->dX));
    CUDA_CHECK(hipFree(ws->dB));
    CUDA_CHECK(hipFree(ws->dA));
    CUSOLVER_CHECK(hipsolverDnDestroy(ws->handle));
    CUDA_CHECK(hipStreamDestroy(ws->stream));
    return 0;
}
namespace {
    char eb[120];
}
const char* get_cuda_error_reason(int n) {
    switch (n) {
    case HX_ALLOCATE: return "HX_ALLOCATE";
    default:
        snprintf(eb, sizeof(eb), "Uknown code %d, add to %s", n, __FILE__);
        return eb;
    }
}


int cuda_solve(int verbose, int n, float* a, float* b, Workspaces* ws) {
    // Numer of right hand sides
    const int nrhs = 1;
    typedef float T;
    int lda=n;
    int ldb=n;
    // copy input data
    size_t dpitch = ws->ldda * sizeof(T);
    size_t spitch = lda * sizeof(T);
    size_t width = n * sizeof(T);
    size_t height = n;
    CUDA_CHECK(hipMemcpy2D(ws->dA, dpitch, a, spitch, width, height,
        hipMemcpyDefault));
    dpitch = ws->lddb * sizeof(T);
    spitch = ldb * sizeof(T);
    height = nrhs;
    CUDA_CHECK(hipMemcpy2D(ws->dB, dpitch, b, spitch, width,height,
        hipMemcpyDefault));
    hipsolverStatus_t gesv_status =
        cusolverDnIRSXgesv(ws->handle, ws->gesv_params, ws->gesv_info, n, nrhs, 
            ws->dA, ws->ldda, ws->dB, ws->lddb, ws->dX, ws->lddx,
                           ws->dwork, ws->dwork_size, &ws->iter, ws->dinfo); // 138 ms
    CUSOLVER_CHECK(gesv_status);
    // check solve status
    int info = 0;
    CUDA_CHECK(
        hipMemcpyAsync(&info, ws->dinfo, 
            sizeof(int), hipMemcpyDeviceToHost, ws->stream));
    CUDA_CHECK(hipStreamSynchronize(ws->stream));
    CUDA_CHECK(hipMemcpy2D(b, n * sizeof(T), ws->dX, 
            ws->lddx * sizeof(T), n * sizeof(T), nrhs,
                            hipMemcpyDefault));
    CUDA_CHECK(hipGetLastError());
    return 0;
}
